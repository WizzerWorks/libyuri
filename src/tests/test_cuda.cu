#include "hip/hip_runtime.h"

#include "catch.hpp"
#include "yuri/cuda/Cuda.h"
#include <numeric>
#include <algorithm>

namespace yuri{

__global__
void iota(int *a)
{
	a[threadIdx.x] = threadIdx.x;
}


__global__
void cpy(const int *a, int* b)
{
	b[threadIdx.x] = a[threadIdx.x];
}

template<typename T>
bool operator==(const std::vector<T>& lhs, const uvector<T>& rhs)
{
	return lhs.size() == rhs.size() &&
			std::equal(lhs.begin(), lhs.end(), rhs.begin());
}




TEST_CASE("cuda") {
	std::vector<int> a(10, 0);
	std::vector<int> b(10, 0);
	uvector<int> c(10);
	std::vector<int> expected(10, 0);
	std::iota(expected.begin(), expected.end(), 0);
	
	auto mem_a = cuda::cuda_alloc<int>(a.size());
	auto mem_b = cuda::cuda_alloc<int>(b.size());
	auto mem_c = cuda::cuda_alloc<int>(c.size());
	dim3 dimBlock( a.size(), 1 );
	dim3 dimGrid( 1, 1 );
			
	SECTION("CUDA COPY") {
		REQUIRE(cuda::copy_to_gpu(mem_a, a));

		iota<<<dimGrid, dimBlock>>>(mem_a.get());
		
		REQUIRE(cuda::copy_to_cpu(a, mem_a));
		
		REQUIRE(a == expected);
		REQUIRE(a != b);
		
		REQUIRE(cuda::copy_to_gpu(mem_b, b));
		
		cpy<<<dimGrid, dimBlock>>>(mem_a.get(), mem_b.get());
		
		REQUIRE(cuda::copy_to_cpu(b, mem_b));
		REQUIRE(a == b);
		
		REQUIRE(cuda::copy_to_gpu(mem_c, c));
				
		cpy<<<dimGrid, dimBlock>>>(mem_b.get(), mem_c.get());
		
		REQUIRE(cuda::copy_to_cpu(c, mem_c));
		
		REQUIRE( a == c );
	}
	
		
}
}
		